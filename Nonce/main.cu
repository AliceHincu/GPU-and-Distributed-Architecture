#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <cstring>

#include "kernel.cuh" 

#define min(a,b)            (((a) < (b)) ? (a) : (b))
#define ZEROS_TO_FIND 2

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// 1. modify path: $env:PATH = "C:\Program Files\Microsoft Visual Studio\2022\Community\VC\Tools\MSVC\14.42.34433\bin\Hostx64\x64;" + $env:PATH
// 2. command to compile:  nvcc -rdc=true main.cu sha1.cu utils.cu kernel.cu  (https://stackoverflow.com/questions/27590166/how-to-compile-multiple-files-in-cuda)
// 3. command to start: ./a
// 4. verify with this link: http://www.sha1-online.com/
int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    //printf("Multiprocessors: %d\n", prop.multiProcessorCount);
    //printf("Max Threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
    //printf("Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
    const int THREADS_PER_BLOCK = min(256, prop.maxThreadsPerBlock); // 256 good default because it balances parallelism and register usage
    const int MAX_BLOCKS = prop.multiProcessorCount * (prop.maxThreadsPerMultiProcessor / THREADS_PER_BLOCK);
    printf("Using MAX_BLOCKS: %d, THREADS_PER_BLOCK: %d\n", MAX_BLOCKS, THREADS_PER_BLOCK);

    const char* h_string = "abc";
    int h_found = 0;
    uint32_t h_nonce = 0;
    int h_string_length = strlen(h_string) + 1;

    uint32_t* d_nonce;
    int* d_found;

    // alocate memory on gpu
    CUDA_CHECK(hipMalloc(&d_nonce, sizeof(uint32_t)));
    CUDA_CHECK(hipMalloc(&d_found, sizeof(int)));
    // copy input to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_const_input), h_string, h_string_length);

    // copy data on gpu
    CUDA_CHECK(hipMemcpy(d_found, &h_found, sizeof(bool), hipMemcpyHostToDevice));

    // start looking for nonce
    find_nonce << <MAX_BLOCKS, THREADS_PER_BLOCK >> > (d_nonce, d_found, ZEROS_TO_FIND);
    CUDA_CHECK(hipDeviceSynchronize());

    // copy resuly on cpu
    CUDA_CHECK(hipMemcpy(&h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&h_nonce, d_nonce, sizeof(uint32_t), hipMemcpyDeviceToHost));

    if (h_found) {
        printf("Nonce found: %I32u\n", h_nonce);
    } else {
        printf("Nonce not found.\n");
    }

    hipFree(d_nonce);
    hipFree(d_found);

    return 0;
}

